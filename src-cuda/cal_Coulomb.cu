#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <stdexcept>
#include "rism3d.h"

void RISM3D :: cal_Coulomb (string esp) {
  __global__ void coulomb(double * de, double * dfr,
			  double4 * dru, double * dqu,
			  double dx, double dy, double dz,
			  int nx, int ny, int nz, int natu);
  __global__ void fk(double2 *, const double4 * __restrict__ , 
		     const double4 * __restrict__ , const double * __restrict__, 
		     int);
  __global__ void beta(double * dfr, double2 * dfk, double ubeta);
  __global__ void beta2(double * de, double ubeta);

  cout << "synthesizing solute Coulomb potential ..." << endl;
  
  hipMalloc(&de, ce -> ngrid * sizeof(double));
  hipMalloc(&dfr, ce -> ngrid * sizeof(double));
  hipMalloc(&dfk, ce -> ngrid * sizeof(double2));
  hipMemset(de, 0.0, ce -> ngrid * sizeof(double));
  hipMemset(dfr, 0.0, ce -> ngrid * sizeof(double));
  hipMemset(dfk, 0.0, ce -> ngrid * sizeof(double2));

  coulomb <<< g, b >>> (de, dfr, su -> dr, su -> dq,
			ce -> dr[0], ce -> dr[1], ce -> dr[2], 
			ce -> grid[0], ce -> grid[1], ce -> grid[2], su -> num);

  fk <<< g, b >>> (dfk, dgv, su -> dr, su -> dq, su -> num);

  double ubeta = hartree2J * bohr / (boltzmann * sv -> temper);
  beta <<< g, b >>> (dfr, dfk, ubeta);

  if (esp.empty()) {
    double ubeta = hartree2J * bohr / (boltzmann * sv -> temper);
    beta2 <<< g, b >>> (de, ubeta);
  } else {
    ifstream in_file;
    in_file.open (esp.c_str());
    double *e = new double[ce -> ngrid];
    double dummy;

    for (int i = 0; i < ce -> ngrid; ++i) {
      string line;
      string data;
      getline(in_file, line);
      stringstream ss(line);
      ss >> setw(20) >> dummy
         >> setw(20) >> dummy
         >> setw(20) >> dummy
         >> setw(20) >> data;
      double evalue;
      try {
        evalue = stod(data);
      } catch (const std::invalid_argument& e) {
        evalue = 0.0;
      }     
      e[i] = evalue;
    }
    in_file.close();

    hipMemcpyAsync(de, e, ce -> ngrid * sizeof(double), hipMemcpyDefault);
    double ubeta = hartree2J / (boltzmann * sv -> temper);
    beta2 <<< g, b >>> (de, ubeta);
    delete[] e;
  }

} 


__global__ void coulomb(double * de, double * dfr,
                        double4 * dru, double * dqu,
                        double bx, double by, double bz,
                        int nx, int ny, int nz, int natu) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  double rx = ((int)threadIdx.x - nx / 2) * bx;
  double ry = ((int)blockIdx.x - ny / 2) * by;
  double rz = ((int)blockIdx.y - nz / 2) * bz;
  for (int iu = 0; iu < natu; ++iu) {
    double delx = rx - dru[iu].x;
    double dely = ry - dru[iu].y;
    double delz = rz - dru[iu].z;
    double ra = sqrt(delx * delx + dely * dely + delz * delz) ;
    if (ra >= 1.0e-5) {
      double qr = dqu[iu] / ra ;
      de[ip] += qr ;
      dfr[ip] += qr * (1 - exp(- ra)) ;
    } else {
      dfr[ip] += dqu[iu] ;
    }
  }
}


__global__ void fk(double2 * dfk, const double4 * __restrict__ dgv, 
		   const double4 * __restrict__ dru, 
		   const double * __restrict__ dqu, int natu) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  double rk2 = dgv[ip].x * dgv[ip].x
    + dgv[ip].y * dgv[ip].y + dgv[ip].z * dgv[ip].z;
  double rk4i = 1.0 / (rk2 * (rk2 + 1.0));
  for (int iu = 0; iu < natu; ++iu) {
    double ruk = dgv[ip].x * dru[iu].x 
      + dgv[ip].y * dru[iu].y + dgv[ip].z * dru[iu].z;
    double tmp = 4.0 * M_PI * dqu[iu] * rk4i;
    dfk[ip].x += tmp * cos(ruk);
    dfk[ip].y -= tmp * sin(ruk);
  }
}


__global__ void beta(double * dfr, double2 * dfk, double ubeta) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  dfr[ip] *= ubeta;
  dfk[ip].x *= ubeta;
  dfk[ip].y *= ubeta;
}

__global__ void beta2(double * de, double ubeta) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  de[ip] *= ubeta;
}
