#include "hip/hip_runtime.h"
#include "solute.h"

void Solute :: init(int n) {
  num = n;
  q = new double[num];
  sig = new double[num];
  eps = new double[num];
  r = new double[num * 4];
}


void Solute :: setup_cuda() {
  hipMalloc(&dq, num * sizeof(double));
  hipMalloc(&dr, num * sizeof(double4));
  hipMemcpyAsync(dq, q, num * sizeof(double), hipMemcpyDefault);
  hipMemcpyAsync(dr, r, num * sizeof(double4), hipMemcpyDefault);
}
